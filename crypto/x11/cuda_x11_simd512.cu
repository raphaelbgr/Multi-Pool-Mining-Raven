#include "hip/hip_runtime.h"
/***************************************************************************************************
 * SIMD512 SM3+ CUDA IMPLEMENTATION (require cuda_x11_simd512_func.cuh)
 */

#include "miner.h"
#include "cuda_helper.h"

#define TPB 128

uint32_t *d_state[MAX_GPUS];
uint4 *d_temp4[MAX_GPUS];

// texture bound to d_temp4[thr_id], for read access in Compaction kernel
texture<uint4, 1, hipReadModeElementType> texRef1D_128;

__constant__ uint8_t c_perm[8][8] =
{
	{ 2, 3, 6, 7, 0, 1, 4, 5 },
	{ 6, 7, 2, 3, 4, 5, 0, 1 },
	{ 7, 6, 5, 4, 3, 2, 1, 0 },
	{ 1, 0, 3, 2, 5, 4, 7, 6 },
	{ 0, 1, 4, 5, 6, 7, 2, 3 },
	{ 6, 7, 2, 3, 0, 1, 4, 5 },
	{ 6, 7, 0, 1, 4, 5, 2, 3 },
	{ 4, 5, 2, 3, 6, 7, 0, 1 }
};

/* used in cuda_x11_simd512_func.cuh (SIMD_Compress2) */
__constant__ uint32_t c_IV_512[32] =
{
	0x0ba16b95, 0x72f999ad, 0x9fecc2ae, 0xba3264fc, 0x5e894929, 0x8e9f30e5, 0x2f1daa37, 0xf0f2c558,
	0xac506643, 0xa90635a5, 0xe25b878b, 0xaab7878f, 0x88817f7a, 0x0a02892b, 0x559a7550, 0x598f657e,
	0x7eef60a1, 0x6b70e3e8, 0x9c1714d1, 0xb958e2a8, 0xab02675e, 0xed1c014f, 0xcd8d65bb, 0xfdb7a257,
	0x09254899, 0xd699c7bc, 0x9019b6dc, 0x2b9022e4, 0x8fa14956, 0x21bf9bd3, 0xb94d0943, 0x6ffddc22
};

__constant__ int c_FFT128_8_16_Twiddle[128] =
{
	1,   1,   1,   1,   1,    1,   1,   1,   1,   1,   1,   1,   1,   1,   1,   1,
	1,  60,   2, 120,   4,  -17,   8, -34,  16, -68,  32, 121,  64, -15, 128, -30,
	1,  46,  60, -67,   2,   92, 120, 123,   4, -73, -17, -11,   8, 111, -34, -22,
	1, -67, 120, -73,   8,  -22, -68, -70,  64,  81, -30, -46,  -2,-123,  17,-111,
	1,-118,  46, -31,  60,  116, -67, -61,   2,  21,  92, -62, 120, -25, 123,-122,
	1, 116,  92,-122, -17,   84, -22,  18,  32, 114, 117, -49, -30, 118,  67,  62,
	1, -31, -67,  21, 120, -122, -73, -50,   8,   9, -22, -89, -68,  52, -70, 114,
	1, -61, 123, -50, -34,   18, -70, -99, 128, -98,  67,  25,  17,  -9,  35, -79
};

__constant__ int c_FFT256_2_128_Twiddle[128] =
{
	  1,  41,-118,  45,  46,  87, -31,  14,
	 60,-110, 116,-127, -67,  80, -61,  69,
	  2,  82,  21,  90,  92, -83, -62,  28,
	120,  37, -25,   3, 123, -97,-122,-119,
	  4, -93,  42, -77, -73,  91,-124,  56,
	-17,  74, -50,   6, -11,  63,  13,  19,
	  8,  71,  84, 103, 111, -75,   9, 112,
	-34,-109,-100,  12, -22, 126,  26,  38,
	 16,-115, -89, -51, -35, 107,  18, -33,
	-68,  39,  57,  24, -44,  -5,  52,  76,
	 32,  27,  79,-102, -70, -43,  36, -66,
	121,  78, 114,  48, -88, -10, 104,-105,
	 64,  54, -99,  53, 117, -86,  72, 125,
	-15,-101, -29,  96,  81, -20, -49,  47,
	128, 108,  59, 106, -23,  85,-113,  -7,
	-30,  55, -58, -65, -95, -40, -98,  94
};

/************* the round function ****************/
#define IF(x, y, z) (((y ^ z) & x) ^ z)
#define MAJ(x, y, z) ((z & y) | ((z | y) & x))

#include "cuda_x11_simd512_func.cuh"

/********************* Message expansion ************************/

/*
 * Reduce modulo 257; result is in [-127; 383]
 * REDUCE(x) := (x&255) - (x>>8)
 */
#define REDUCE(x) (((x)&255) - ((x)>>8))

/*
 * Reduce from [-127; 383] to [-128; 128]
 * EXTRA_REDUCE_S(x) := x<=128 ? x : x-257
 */
#define EXTRA_REDUCE_S(x) ((x)<=128 ? (x) : (x)-257)

/*
 * Reduce modulo 257; result is in [-128; 128]
 */
#define REDUCE_FULL_S(x) EXTRA_REDUCE_S(REDUCE(x))

// Parallelization:
//
// FFT_8  wird 2 times 8-fach parallel ausgeführt (in FFT_64)
//        and  1 time 16-fach parallel (in FFT_128_full)
//
// STEP8_IF and STEP8_MAJ beinhalten je 2x 8-fach parallel Operations

/**
 * FFT_8 using w=4 as 8th root of unity
 * Unrolled decimation in frequency (DIF) radix-2 NTT.
 * Output data is in revbin_permuted order.
 */
static __device__ __forceinline__
void FFT_8(int *y, int stripe)
{
#define X(i) y[stripe*i]

#define DO_REDUCE(i) \
	X(i) = REDUCE(X(i))

#define DO_REDUCE_FULL_S(i) \
do { \
	X(i) = REDUCE(X(i)); \
	X(i) = EXTRA_REDUCE_S(X(i)); \
} while(0)

#define BUTTERFLY(i,j,n) \
do { \
	int u = X(i); \
	X(i) += X(j); \
	X(j) = (u - X(j)) << (2 * n); \
} while(0)

	BUTTERFLY(0, 4, 0);
	BUTTERFLY(1, 5, 1);
	BUTTERFLY(2, 6, 2);
	BUTTERFLY(3, 7, 3);

	DO_REDUCE(6);
	DO_REDUCE(7);

	BUTTERFLY(0, 2, 0);
	BUTTERFLY(4, 6, 0);
	BUTTERFLY(1, 3, 2);
	BUTTERFLY(5, 7, 2);

	DO_REDUCE(7);

	BUTTERFLY(0, 1, 0);
	BUTTERFLY(2, 3, 0);
	BUTTERFLY(4, 5, 0);
	BUTTERFLY(6, 7, 0);

	DO_REDUCE_FULL_S(0);
	DO_REDUCE_FULL_S(1);
	DO_REDUCE_FULL_S(2);
	DO_REDUCE_FULL_S(3);
	DO_REDUCE_FULL_S(4);
	DO_REDUCE_FULL_S(5);
	DO_REDUCE_FULL_S(6);
	DO_REDUCE_FULL_S(7);

#undef X
#undef DO_REDUCE
#undef DO_REDUCE_FULL_S
#undef BUTTERFLY
}

/**
 * FFT_16 using w=2 as 16th root of unity
 * Unrolled decimation in frequency (DIF) radix-2 NTT.
 * Output data is in revbin_permuted order.
 */
static __device__ __forceinline__
void FFT_16(int *y)
{
#define DO_REDUCE_FULL_S(i) \
	do { \
		y[i] = REDUCE(y[i]); \
		y[i] = EXTRA_REDUCE_S(y[i]); \
	} while(0)

	int u,v;

	u = y[0];
	y[0] += y[1];
	y[1] = (u - y[1]) << (threadIdx.x&7);

	if ((threadIdx.x&7) >=3)
		y[1] = REDUCE(y[1]);  // 11...15

	u = SHFL((int)y[0],  (threadIdx.x&3),8); // 0,1,2,3  0,1,2,3
	v = SHFL((int)y[0],4+(threadIdx.x&3),8); // 4,5,6,7  4,5,6,7
	y[0] = ((threadIdx.x&7) < 4) ? (u+v) : ((u-v) << (2*(threadIdx.x&3)));

	u = SHFL((int)y[1],  (threadIdx.x&3),8); // 8,9,10,11    8,9,10,11
	v = SHFL((int)y[1],4+(threadIdx.x&3),8); // 12,13,14,15  12,13,14,15
	y[1] = ((threadIdx.x&7) < 4) ? (u+v) : ((u-v) << (2*(threadIdx.x&3)));


	if ((threadIdx.x&1) && (threadIdx.x&7) >= 4)
	{
		y[0] = REDUCE(y[0]);  // 5, 7
		y[1] = REDUCE(y[1]);  // 13, 15
	}

	u = SHFL((int)y[0],  (threadIdx.x&5),8); // 0,1,0,1  4,5,4,5
	v = SHFL((int)y[0],2+(threadIdx.x&5),8); // 2,3,2,3  6,7,6,7
	y[0] = ((threadIdx.x&3) < 2) ? (u+v) : ((u-v) << (4*(threadIdx.x&1)));

	u = SHFL((int)y[1],  (threadIdx.x&5),8); // 8,9,8,9      12,13,12,13
	v = SHFL((int)y[1],2+(threadIdx.x&5),8); // 10,11,10,11  14,15,14,15
	y[1] = ((threadIdx.x&3) < 2) ? (u+v) : ((u-v) << (4*(threadIdx.x&1)));

	u = SHFL((int)y[0],  (threadIdx.x&6),8); // 0,0,2,2      4,4,6,6
	v = SHFL((int)y[0],1+(threadIdx.x&6),8); // 1,1,3,3      5,5,7,7
	y[0] = ((threadIdx.x&1) < 1) ? (u+v) : (u-v);

	u = SHFL((int)y[1],  (threadIdx.x&6),8); // 8,8,10,10    12,12,14,14
	v = SHFL((int)y[1],1+(threadIdx.x&6),8); // 9,9,11,11    13,13,15,15
	y[1] = ((threadIdx.x&1) < 1) ? (u+v) : (u-v);

	DO_REDUCE_FULL_S( 0); // 0...7
	DO_REDUCE_FULL_S( 1); // 8...15

#undef DO_REDUCE_FULL_S
}

static __device__ __forceinline__
void FFT_128_full(int y[128])
{
	int i;

	FFT_8(y+0,2); // eight parallel FFT8's
	FFT_8(y+1,2); // eight parallel FFT8's

#pragma unroll 16
	for (i=0; i<16; i++)
	/*if (i & 7)*/ y[i] = REDUCE(y[i]*c_FFT128_8_16_Twiddle[i*8+(threadIdx.x&7)]);

#pragma unroll 8
	for (i=0; i<16; i+=2)
		FFT_16(y + i);  // eight sequential FFT16's, each one executed in parallel by 8 threads
}

static __device__ __forceinline__
void FFT_256_halfzero(int y[256])
{
	/*
	 * FFT_256 using w=41 as 256th root of unity.
	 * Decimation in frequency (DIF) NTT.
	 * Output data is in revbin_permuted order.
	 * In place.
	 */
	const int tmp = y[15];

#pragma unroll 8
	for (int i=0; i<8; i++)
		y[16+i] = REDUCE(y[i] * c_FFT256_2_128_Twiddle[8*i+(threadIdx.x&7)]);
#pragma unroll 8
	for (int i=24; i<32; i++)
		y[i] = 0;

	/* handle X^255 with an additional butterfly */
	if ((threadIdx.x&7) == 7)
	{
		y[15] = REDUCE(tmp + 1);
		y[31] = REDUCE((tmp - 1) * c_FFT256_2_128_Twiddle[127]);
	}

	FFT_128_full(y);
	FFT_128_full(y+16);
}

/***************************************************/

static __device__ __forceinline__
void Expansion(const uint32_t *data, uint4 *g_temp4)
{
	/* Message Expansion using Number Theoretical Transform similar to FFT */
	int expanded[32];
#pragma unroll 4
	for (int i=0; i < 4; i++) {
		expanded[  i] = __byte_perm(SHFL((int)data[0], 2*i, 8), SHFL((int)data[0], (2*i)+1, 8), threadIdx.x&7)&0xff;
		expanded[4+i] = __byte_perm(SHFL((int)data[1], 2*i, 8), SHFL((int)data[1], (2*i)+1, 8), threadIdx.x&7)&0xff;
	}
#pragma unroll 8
	for (int i=8; i < 16; i++)
		expanded[i] = 0;

	FFT_256_halfzero(expanded);

	// store w matrices in global memory

#define mul_185(x) ( (x)*185 )
#define mul_233(x) ( (x)*233 )

	uint4 vec0;
	int P, Q, P1, Q1, P2, Q2;
	bool even = (threadIdx.x & 1) == 0;

	P1 = expanded[ 0]; P2 = SHFL(expanded[ 2], (threadIdx.x-1)&7, 8); P = even ? P1 : P2;
	Q1 = expanded[16]; Q2 = SHFL(expanded[18], (threadIdx.x-1)&7, 8); Q = even ? Q1 : Q2;
	vec0.x = SHFL((int)__byte_perm(mul_185(P),  mul_185(Q) , 0x5410), c_perm[0][threadIdx.x&7], 8);
	P1 = expanded[ 8]; P2 = SHFL(expanded[10], (threadIdx.x-1)&7, 8); P = even ? P1 : P2;
	Q1 = expanded[24]; Q2 = SHFL(expanded[26], (threadIdx.x-1)&7, 8); Q = even ? Q1 : Q2;
	vec0.y = SHFL((int)__byte_perm(mul_185(P),  mul_185(Q) , 0x5410), c_perm[0][threadIdx.x&7], 8);
	P1 = expanded[ 4]; P2 = SHFL(expanded[ 6], (threadIdx.x-1)&7, 8); P = even ? P1 : P2;
	Q1 = expanded[20]; Q2 = SHFL(expanded[22], (threadIdx.x-1)&7, 8); Q = even ? Q1 : Q2;
	vec0.z = SHFL((int)__byte_perm(mul_185(P),  mul_185(Q) , 0x5410), c_perm[0][threadIdx.x&7], 8);
	P1 = expanded[12]; P2 = SHFL(expanded[14], (threadIdx.x-1)&7, 8); P = even ? P1 : P2;
	Q1 = expanded[28]; Q2 = SHFL(expanded[30], (threadIdx.x-1)&7, 8); Q = even ? Q1 : Q2;
	vec0.w = SHFL((int)__byte_perm(mul_185(P),  mul_185(Q) , 0x5410), c_perm[0][threadIdx.x&7], 8);
	g_temp4[threadIdx.x&7] = vec0;

	P1 = expanded[ 1]; P2 = SHFL(expanded[ 3], (threadIdx.x-1)&7, 8); P = even ? P1 : P2;
	Q1 = expanded[17]; Q2 = SHFL(expanded[19], (threadIdx.x-1)&7, 8); Q = even ? Q1 : Q2;
	vec0.x = SHFL((int)__byte_perm(mul_185(P),  mul_185(Q) , 0x5410), c_perm[1][threadIdx.x&7], 8);
	P1 = expanded[ 9]; P2 = SHFL(expanded[11], (threadIdx.x-1)&7, 8); P = even ? P1 : P2;
	Q1 = expanded[25]; Q2 = SHFL(expanded[27], (threadIdx.x-1)&7, 8); Q = even ? Q1 : Q2;
	vec0.y = SHFL((int)__byte_perm(mul_185(P),  mul_185(Q) , 0x5410), c_perm[1][threadIdx.x&7], 8);
	P1 = expanded[ 5]; P2 = SHFL(expanded[ 7], (threadIdx.x-1)&7, 8); P = even ? P1 : P2;
	Q1 = expanded[21]; Q2 = SHFL(expanded[23], (threadIdx.x-1)&7, 8); Q = even ? Q1 : Q2;
	vec0.z = SHFL((int)__byte_perm(mul_185(P),  mul_185(Q) , 0x5410), c_perm[1][threadIdx.x&7], 8);
	P1 = expanded[13]; P2 = SHFL(expanded[15], (threadIdx.x-1)&7, 8); P = even ? P1 : P2;
	Q1 = expanded[29]; Q2 = SHFL(expanded[31], (threadIdx.x-1)&7, 8); Q = even ? Q1 : Q2;
	vec0.w = SHFL((int)__byte_perm(mul_185(P),  mul_185(Q) , 0x5410), c_perm[1][threadIdx.x&7], 8);
	g_temp4[8+(threadIdx.x&7)] = vec0;

	bool hi = (threadIdx.x&7)>=4;

	P1 = hi?expanded[ 1]:expanded[ 0]; P2 = SHFL(hi?expanded[ 3]:expanded[ 2], (threadIdx.x+1)&7, 8); P = !even ? P1 : P2;
	Q1 = hi?expanded[17]:expanded[16]; Q2 = SHFL(hi?expanded[19]:expanded[18], (threadIdx.x+1)&7, 8); Q = !even ? Q1 : Q2;
	vec0.x = SHFL((int)__byte_perm(mul_185(P),  mul_185(Q) , 0x5410), c_perm[2][threadIdx.x&7], 8);
	P1 = hi?expanded[ 9]:expanded[ 8]; P2 = SHFL(hi?expanded[11]:expanded[10], (threadIdx.x+1)&7, 8); P = !even ? P1 : P2;
	Q1 = hi?expanded[25]:expanded[24]; Q2 = SHFL(hi?expanded[27]:expanded[26], (threadIdx.x+1)&7, 8); Q = !even ? Q1 : Q2;
	vec0.y = SHFL((int)__byte_perm(mul_185(P),  mul_185(Q) , 0x5410), c_perm[2][threadIdx.x&7], 8);
	P1 = hi?expanded[ 5]:expanded[ 4]; P2 = SHFL(hi?expanded[ 7]:expanded[ 6], (threadIdx.x+1)&7, 8); P = !even ? P1 : P2;
	Q1 = hi?expanded[21]:expanded[20]; Q2 = SHFL(hi?expanded[23]:expanded[22], (threadIdx.x+1)&7, 8); Q = !even ? Q1 : Q2;
	vec0.z = SHFL((int)__byte_perm(mul_185(P),  mul_185(Q) , 0x5410), c_perm[2][threadIdx.x&7], 8);
	P1 = hi?expanded[13]:expanded[12]; P2 = SHFL(hi?expanded[15]:expanded[14], (threadIdx.x+1)&7, 8); P = !even ? P1 : P2;
	Q1 = hi?expanded[29]:expanded[28]; Q2 = SHFL(hi?expanded[31]:expanded[30], (threadIdx.x+1)&7, 8); Q = !even ? Q1 : Q2;
	vec0.w = SHFL((int)__byte_perm(mul_185(P),  mul_185(Q) , 0x5410), c_perm[2][threadIdx.x&7], 8);
	g_temp4[16+(threadIdx.x&7)] = vec0;

	bool lo = (threadIdx.x&7)<4;

	P1 = lo?expanded[ 1]:expanded[ 0]; P2 = SHFL(lo?expanded[ 3]:expanded[ 2], (threadIdx.x+1)&7, 8); P = !even ? P1 : P2;
	Q1 = lo?expanded[17]:expanded[16]; Q2 = SHFL(lo?expanded[19]:expanded[18], (threadIdx.x+1)&7, 8); Q = !even ? Q1 : Q2;
	vec0.x = SHFL((int)__byte_perm(mul_185(P),  mul_185(Q) , 0x5410), c_perm[3][threadIdx.x&7], 8);
	P1 = lo?expanded[ 9]:expanded[ 8]; P2 = SHFL(lo?expanded[11]:expanded[10], (threadIdx.x+1)&7, 8); P = !even ? P1 : P2;
	Q1 = lo?expanded[25]:expanded[24]; Q2 = SHFL(lo?expanded[27]:expanded[26], (threadIdx.x+1)&7, 8); Q = !even ? Q1 : Q2;
	vec0.y = SHFL((int)__byte_perm(mul_185(P),  mul_185(Q) , 0x5410), c_perm[3][threadIdx.x&7], 8);
	P1 = lo?expanded[ 5]:expanded[ 4]; P2 = SHFL(lo?expanded[ 7]:expanded[ 6], (threadIdx.x+1)&7, 8); P = !even ? P1 : P2;
	Q1 = lo?expanded[21]:expanded[20]; Q2 = SHFL(lo?expanded[23]:expanded[22], (threadIdx.x+1)&7, 8); Q = !even ? Q1 : Q2;
	vec0.z = SHFL((int)__byte_perm(mul_185(P),  mul_185(Q) , 0x5410), c_perm[3][threadIdx.x&7], 8);
	P1 = lo?expanded[13]:expanded[12]; P2 = SHFL(lo?expanded[15]:expanded[14], (threadIdx.x+1)&7, 8); P = !even ? P1 : P2;
	Q1 = lo?expanded[29]:expanded[28]; Q2 = SHFL(lo?expanded[31]:expanded[30], (threadIdx.x+1)&7, 8); Q = !even ? Q1 : Q2;
	vec0.w = SHFL((int)__byte_perm(mul_185(P),  mul_185(Q) , 0x5410), c_perm[3][threadIdx.x&7], 8);
	g_temp4[24+(threadIdx.x&7)] = vec0;

	bool sel = ((threadIdx.x+2)&7) >= 4;  // 2,3,4,5

	P1 = sel?expanded[0]:expanded[1]; Q1 = SHFL(P1, threadIdx.x^1, 8);
	Q2 = sel?expanded[2]:expanded[3]; P2 = SHFL(Q2, threadIdx.x^1, 8);
	P = even? P1 : P2; Q = even? Q1 : Q2;
	vec0.x = SHFL((int)__byte_perm(mul_233(P),  mul_233(Q) , 0x5410), c_perm[4][threadIdx.x&7], 8);
	P1 = sel?expanded[8]:expanded[9]; Q1 = SHFL(P1, threadIdx.x^1, 8);
	Q2 = sel?expanded[10]:expanded[11]; P2 = SHFL(Q2, threadIdx.x^1, 8);
	P = even? P1 : P2; Q = even? Q1 : Q2;
	vec0.y = SHFL((int)__byte_perm(mul_233(P),  mul_233(Q) , 0x5410), c_perm[4][threadIdx.x&7], 8);
	P1 = sel?expanded[4]:expanded[5]; Q1 = SHFL(P1, threadIdx.x^1, 8);
	Q2 = sel?expanded[6]:expanded[7]; P2 = SHFL(Q2, threadIdx.x^1, 8);
	P = even? P1 : P2; Q = even? Q1 : Q2;
	vec0.z = SHFL((int)__byte_perm(mul_233(P),  mul_233(Q) , 0x5410), c_perm[4][threadIdx.x&7], 8);
	P1 = sel?expanded[12]:expanded[13]; Q1 = SHFL(P1, threadIdx.x^1, 8);
	Q2 = sel?expanded[14]:expanded[15]; P2 = SHFL(Q2, threadIdx.x^1, 8);
	P = even? P1 : P2; Q = even? Q1 : Q2;
	vec0.w = SHFL((int)__byte_perm(mul_233(P),  mul_233(Q) , 0x5410), c_perm[4][threadIdx.x&7], 8);

	g_temp4[32+(threadIdx.x&7)] = vec0;

	P1 = sel?expanded[1]:expanded[0]; Q1 = SHFL(P1, threadIdx.x^1, 8);
	Q2 = sel?expanded[3]:expanded[2]; P2 = SHFL(Q2, threadIdx.x^1, 8);
	P = even? P1 : P2; Q = even? Q1 : Q2;
	vec0.x = SHFL((int)__byte_perm(mul_233(P),  mul_233(Q) , 0x5410), c_perm[5][threadIdx.x&7], 8);
	P1 = sel?expanded[9]:expanded[8]; Q1 = SHFL(P1, threadIdx.x^1, 8);
	Q2 = sel?expanded[11]:expanded[10]; P2 = SHFL(Q2, threadIdx.x^1, 8);
	P = even? P1 : P2; Q = even? Q1 : Q2;
	vec0.y = SHFL((int)__byte_perm(mul_233(P),  mul_233(Q) , 0x5410), c_perm[5][threadIdx.x&7], 8);
	P1 = sel?expanded[5]:expanded[4]; Q1 = SHFL(P1, threadIdx.x^1, 8);
	Q2 = sel?expanded[7]:expanded[6]; P2 = SHFL(Q2, threadIdx.x^1, 8);
	P = even? P1 : P2; Q = even? Q1 : Q2;
	vec0.z = SHFL((int)__byte_perm(mul_233(P),  mul_233(Q) , 0x5410), c_perm[5][threadIdx.x&7], 8);
	P1 = sel?expanded[13]:expanded[12]; Q1 = SHFL(P1, threadIdx.x^1, 8);
	Q2 = sel?expanded[15]:expanded[14]; P2 = SHFL(Q2, threadIdx.x^1, 8);
	P = even? P1 : P2; Q = even? Q1 : Q2;
	vec0.w = SHFL((int)__byte_perm(mul_233(P),  mul_233(Q) , 0x5410), c_perm[5][threadIdx.x&7], 8);

	g_temp4[40+(threadIdx.x&7)] = vec0;

	int t;
	t = SHFL(expanded[17],(threadIdx.x+4)&7,8); P1 = sel?t:expanded[16]; Q1 = SHFL(P1, threadIdx.x^1, 8);
	t = SHFL(expanded[19],(threadIdx.x+4)&7,8); Q2 = sel?t:expanded[18]; P2 = SHFL(Q2, threadIdx.x^1, 8);
	P = even? P1 : P2; Q = even? Q1 : Q2;
	vec0.x = SHFL((int)__byte_perm(mul_233(P),  mul_233(Q) , 0x5410), c_perm[6][threadIdx.x&7], 8);
	t = SHFL(expanded[25],(threadIdx.x+4)&7,8); P1 = sel?t:expanded[24]; Q1 = SHFL(P1, threadIdx.x^1, 8);
	t = SHFL(expanded[27],(threadIdx.x+4)&7,8); Q2 = sel?t:expanded[26]; P2 = SHFL(Q2, threadIdx.x^1, 8);
	P = even? P1 : P2; Q = even? Q1 : Q2;
	vec0.y = SHFL((int)__byte_perm(mul_233(P),  mul_233(Q) , 0x5410), c_perm[6][threadIdx.x&7], 8);
	t = SHFL(expanded[21],(threadIdx.x+4)&7,8); P1 = sel?t:expanded[20]; Q1 = SHFL(P1, threadIdx.x^1, 8);
	t = SHFL(expanded[23],(threadIdx.x+4)&7,8); Q2 = sel?t:expanded[22]; P2 = SHFL(Q2, threadIdx.x^1, 8);
	P = even? P1 : P2; Q = even? Q1 : Q2;
	vec0.z = SHFL((int)__byte_perm(mul_233(P),  mul_233(Q) , 0x5410), c_perm[6][threadIdx.x&7], 8);
	t = SHFL(expanded[29],(threadIdx.x+4)&7,8); P1 = sel?t:expanded[28]; Q1 = SHFL(P1, threadIdx.x^1, 8);
	t = SHFL(expanded[31],(threadIdx.x+4)&7,8); Q2 = sel?t:expanded[30]; P2 = SHFL(Q2, threadIdx.x^1, 8);
	P = even? P1 : P2; Q = even? Q1 : Q2;
	vec0.w = SHFL((int)__byte_perm(mul_233(P),  mul_233(Q) , 0x5410), c_perm[6][threadIdx.x&7], 8);

	g_temp4[48+(threadIdx.x&7)] = vec0;

	t = SHFL(expanded[16],(threadIdx.x+4)&7,8); P1 = sel?expanded[17]:t; Q1 = SHFL(P1, threadIdx.x^1, 8);
	t = SHFL(expanded[18],(threadIdx.x+4)&7,8); Q2 = sel?expanded[19]:t; P2 = SHFL(Q2, threadIdx.x^1, 8);
	P = even? P1 : P2; Q = even? Q1 : Q2;
	vec0.x = SHFL((int)__byte_perm(mul_233(P),  mul_233(Q) , 0x5410), c_perm[7][threadIdx.x&7], 8);
	t = SHFL(expanded[24],(threadIdx.x+4)&7,8); P1 = sel?expanded[25]:t; Q1 = SHFL(P1, threadIdx.x^1, 8);
	t = SHFL(expanded[26],(threadIdx.x+4)&7,8); Q2 = sel?expanded[27]:t; P2 = SHFL(Q2, threadIdx.x^1, 8);
	P = even? P1 : P2; Q = even? Q1 : Q2;
	vec0.y = SHFL((int)__byte_perm(mul_233(P),  mul_233(Q) , 0x5410), c_perm[7][threadIdx.x&7], 8);
	t = SHFL(expanded[20],(threadIdx.x+4)&7,8); P1 = sel?expanded[21]:t; Q1 = SHFL(P1, threadIdx.x^1, 8);
	t = SHFL(expanded[22],(threadIdx.x+4)&7,8); Q2 = sel?expanded[23]:t; P2 = SHFL(Q2, threadIdx.x^1, 8);
	P = even? P1 : P2; Q = even? Q1 : Q2;
	vec0.z = SHFL((int)__byte_perm(mul_233(P),  mul_233(Q) , 0x5410), c_perm[7][threadIdx.x&7], 8);
	t = SHFL(expanded[28],(threadIdx.x+4)&7,8); P1 = sel?expanded[29]:t; Q1 = SHFL(P1, threadIdx.x^1, 8);
	t = SHFL(expanded[30],(threadIdx.x+4)&7,8); Q2 = sel?expanded[31]:t; P2 = SHFL(Q2, threadIdx.x^1, 8);
	P = even? P1 : P2; Q = even? Q1 : Q2;
	vec0.w = SHFL((int)__byte_perm(mul_233(P),  mul_233(Q) , 0x5410), c_perm[7][threadIdx.x&7], 8);

	g_temp4[56+(threadIdx.x&7)] = vec0;

#undef mul_185
#undef mul_233
}

/***************************************************/

__global__ __launch_bounds__(TPB, 4)
void x11_simd512_gpu_expand_64(uint32_t threads, uint32_t *g_hash, uint4 *g_temp4)
{
	int threadBloc = (blockDim.x * blockIdx.x + threadIdx.x) / 8;
	if (threadBloc < threads)
	{
		int hashPosition = threadBloc * 16;
		uint32_t *inpHash = &g_hash[hashPosition];

		// Read hash per 8 threads
		uint32_t Hash[2];
		int ndx = threadIdx.x & 7;
		Hash[0] = inpHash[ndx];
		Hash[1] = inpHash[ndx + 8];

		// Puffer für expandierte Nachricht
		uint4 *temp4 = &g_temp4[hashPosition * 4];

		Expansion(Hash, temp4);
	}
}

__global__ __launch_bounds__(TPB, 1)
void x11_simd512_gpu_compress1_64(uint32_t threads, uint32_t *g_hash, uint4 *g_fft4, uint32_t *g_state)
{
	uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
	if (thread < threads)
	{
		uint32_t *Hash = &g_hash[thread * 16];
		Compression1(Hash, thread, g_fft4, g_state);
	}
}

__global__ __launch_bounds__(TPB, 1)
void x11_simd512_gpu_compress2_64(uint32_t threads, uint4 *g_fft4, uint32_t *g_state)
{
	uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
	if (thread < threads)
	{
		Compression2(thread, g_fft4, g_state);
	}
}

__global__ __launch_bounds__(TPB, 2)
void x11_simd512_gpu_compress_64_maxwell(uint32_t threads, uint32_t *g_hash, uint4 *g_fft4, uint32_t *g_state)
{
	uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
	if (thread < threads)
	{
		uint32_t *Hash = &g_hash[thread * 16];
		Compression1(Hash, thread, g_fft4, g_state);
		Compression2(thread, g_fft4, g_state);
	}
}

__global__ __launch_bounds__(TPB, 2)
void x11_simd512_gpu_final_64(uint32_t threads, uint32_t *g_hash, uint4 *g_fft4, uint32_t *g_state)
{
	uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
	if (thread < threads)
	{
		uint32_t *Hash = &g_hash[thread * 16];
		Final(Hash, thread, g_fft4, g_state);
	}
}


__host__
int x11_simd512_cpu_init(int thr_id, uint32_t threads)
{
	size_t temp4size = sizeof(uint4) * 64 * threads;
	CUDA_SAFE_CALL(hipMalloc(&d_temp4[thr_id], temp4size));
	CUDA_SAFE_CALL(hipMalloc(&d_state[thr_id], sizeof(int) * 32 * threads));

	// Texture for 128-Bit Zugriffe
	hipChannelFormatDesc channelDesc128 = hipCreateChannelDesc<uint4>();
	texRef1D_128.normalized = 0;
	texRef1D_128.filterMode = hipFilterModePoint;
	texRef1D_128.addressMode[0] = hipAddressModeClamp;

	CUDA_SAFE_CALL(hipBindTexture(0, &texRef1D_128, d_temp4[thr_id], &channelDesc128, temp4size));

	return 0;
}

__host__
void x11_simd512_cpu_free(int thr_id)
{
	int dev_id = device_map[thr_id];
	if (device_sm[dev_id] >= 300 && cuda_arch[dev_id] >= 300) {
		hipFree(d_temp4[thr_id]);
		hipFree(d_state[thr_id]);
	}
}

__host__
void x11_simd512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_hash)
{
	const uint32_t threadsperblock = TPB;
	int dev_id = device_map[thr_id];

	dim3 block(threadsperblock);
	dim3 grid((threads + threadsperblock-1) / threadsperblock);
	dim3 gridX8(grid.x * 8);

	x11_simd512_gpu_expand_64 <<<gridX8, block>>> (threads, d_hash, d_temp4[thr_id]);
	CUDA_SAFE_CALL(hipGetLastError());

	if (device_sm[dev_id] >= 500 && cuda_arch[dev_id] >= 500)
	{
		x11_simd512_gpu_compress_64_maxwell <<< grid, block, 0, gpustream[thr_id] >>> (threads, d_hash, d_temp4[thr_id], d_state[thr_id]);
		CUDA_SAFE_CALL(hipGetLastError());
	}
	else
	{
		x11_simd512_gpu_compress1_64 <<< grid, block, 0, gpustream[thr_id] >>> (threads, d_hash, d_temp4[thr_id], d_state[thr_id]);
		CUDA_SAFE_CALL(hipGetLastError());
		x11_simd512_gpu_compress2_64 <<< grid, block, 0, gpustream[thr_id] >>> (threads, d_temp4[thr_id], d_state[thr_id]);
		CUDA_SAFE_CALL(hipGetLastError());
	}

	x11_simd512_gpu_final_64 <<<grid, block, 0, gpustream[thr_id] >>> (threads, d_hash, d_temp4[thr_id], d_state[thr_id]);
	CUDA_SAFE_CALL(hipGetLastError());

	//MyStreamSynchronize(NULL, order, thr_id);
}
